#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

#if SHARED == 0
    #define ACC(x,y,z) sharedMemAcc(x,y,z)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
#endif

//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
const float planetMass = 3e8;
const __device__ float starMass = 5e10;

const float scene_scale = 2e2; //size of the height map in simulation space

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;
glm::vec3 * dev_acc;

int iteration = 1;

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__
unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ 
glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Generate randomized starting positions for the planets in the XY plane
//Also initialized the masses
__global__
void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = rand.z;
        
		arr[index].w = mass;
    }
}

//Determine velocity from the distance from the center star. Not super physically accurate because 
//the mass ratio is too close, but it makes for an interesting looking scene
__global__
void generateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec4 * pos)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G*starMass/r);
        glm::vec3 D = glm::normalize(glm::cross(R/r,glm::vec3(0,0,1)));
        arr[index].x = s*D.x;
        arr[index].y = s*D.y;
        arr[index].z = s*D.z;
    }
}

//Generate randomized starting velocities in the XY plane
__global__
void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index) - 0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = rand.z;
    }
}

//generate random point on a sphere
__device__ glm::vec3 getRandomPointOnSphere(float randomSeed, int time){
	
	thrust::default_random_engine rng(hash(randomSeed*time));
	thrust::uniform_real_distribution<float> u01(-1, 1);
    thrust::uniform_real_distribution<float> u02(0, 2*PI);

    glm::vec3 point (0.5f, 0.5f, 0.5f);
        
    float z = (float)u01(rng);
    float theta = (float)u02(rng);

    point.x = sqrt(1 - (z*z)) * cos(theta);
    point.y = sqrt( 1 - (z*z)) * sin(theta);
    point.z = z;
	
	return point;
}


__device__
glm::vec3 wander(glm::vec3 vel, glm::vec4 pos, int index, float dt, int time){
	
	//find random displacement on sphere
	glm::vec3 vWander = getRandomPointOnSphere(index, time);
	
	//translate to new center
	glm::vec3 myPos(pos.x, pos.y, pos.z);
	glm::vec3 center = myPos + vel;
	vWander += center;

	vel = VRADIUS*glm::normalize(vWander);
	
	return vel;
}

__device__
glm::vec3 alignment(int N, glm::vec4 myPos, glm::vec3 myVel, glm::vec3* vel, glm::vec4* pos){

	int numTiles = ceil((float)N/blockSize);
	__shared__ glm::vec4 posTile[blockSize];
	__shared__ glm::vec3 velTile[blockSize];

	glm::vec3 vAlign(0);
	int numNeigbors = 0;

	for(int i = 0; i < numTiles; ++i){
	
		//load positions and velocities to shared memory
		posTile[threadIdx.x] = pos[i*blockSize + threadIdx.x];
		velTile[threadIdx.x] = vel[i*blockSize + threadIdx.x];
		__syncthreads();

		//find weighted average of velocities in neighborhood
		for( int j = 0; j < blockSize; ++j){
			if(blockSize*i+j < N){
				//check if in neigborhood
				float dist = glm::length(posTile[threadIdx.x] - myPos);
				if(dist < NEIGHBOR_RAD){
					vAlign += velTile[threadIdx.x];
					++numNeigbors;
				}
			}
		}
		__syncthreads();

	}

	return glm::normalize(vAlign*(1.0f/numNeigbors));

}

//updates velocity
__global__
void updateF(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc, int time)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    glm::vec4 my_pos;
    glm::vec3 my_vel;

	if(index < N){

		my_pos = pos[index];
		my_vel = vel[index];
		//vel[index] += wander(vel[index], my_pos, index, dt, time);
		vel[index] = alignment(N, my_pos, my_vel, vel, pos); 
	}
}

//does euler integration to find new position
__global__
void updateS(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;

		//pos[index].x += 0.01;
  //      pos[index].y += 0.01;
  //      pos[index].z += 0.01;

    }
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__
void sendToVBO(int N, glm::vec4 * pos, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;
	float c_scale_b = -2.0f / s_scale;

    if(index<N)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
		vbo[4*index+2] = pos[index].z*c_scale_b;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__
void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N)
{
    numObjects = N;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize)));

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_acc, N*sizeof(glm::vec3)); 
    checkCUDAErrorWithLine("Kernel failed!");

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");
    
	generateRandomVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, scene_scale);
	checkCUDAErrorWithLine("Kernel failed!");

	//generateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
    //checkCUDAErrorWithLine("Kernel failed!");
    hipDeviceSynchronize();
}

void cudaNBodyUpdateWrapper(float dt)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_pos, dev_vel, dev_acc, iteration);
    checkCUDAErrorWithLine("Kernel failed!");
    updateS<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
    checkCUDAErrorWithLine("Kernel failed!");
    hipDeviceSynchronize();
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
	iteration ++;
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
	sendToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}
