#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

#if SHARED == 1
    #define ACC(x,y,z) sharedMemAcc(x,y,z)
	#define FLOCK(x,y,z,v) sharedMemFlocking(x,y,z,v)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
	#define FLOCK(x,y,z,v) navieFlocking(x,y,z,v)
#endif



//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
int numPredator;
const float planetMass = 3e8;
const __device__ float starMass = 5e10;

const float scene_scale = 2e2; //size of the height map in simulation space

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;

glm::vec4 * pre_pos;
glm::vec3 * pre_vel;

glm::vec3 vWander;
float kWander = 1.0f;

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__
unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ 
glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}


__global__
void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale * (generateRandomNumberFromThread(time + index, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = rand.z;
        arr[index].w = mass;
    }
}


//Generate randomized starting velocities in the XYZ plane
__global__
void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index*2 + 1));
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = rand.z;
    }
}


__device__
glm::vec3 calAlignment(glm::vec3 vel)
{
	return vel;
}

__device__ 
glm::vec3 calSeparation(glm::vec4 d)
{
	glm::vec3 dir;
	dir.x = d.x;
	dir.y = d.y;
	dir.z = d.z;

	float len = dir.x*dir.x + dir.y*dir.y + dir.z*dir.z;
	if(len > DBL_EPSILON)
		return dir/len;
	else
		return glm::vec3(0);
}

__device__
glm::vec3 calCohesion(glm::vec4 them)
{
	return glm::vec3(them);
}


__device__
glm::vec3 navieFlocking(int N, glm::vec4 my_pos, glm::vec4* their_pos, glm::vec3* vel)
{
	glm::vec3 align, separa, cohes;
	int count = 0;
	float totalDist = 0.f;
	float totalMass = 0.f;
	for(int i = 0; i < N; i++)
	{
		glm::vec4 delta = my_pos - their_pos[i];
		float dist = sqrtf(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
		if(dist < RNEIGHBOR)
		{
			count ++;
			align += calAlignment(vel[i]) * dist;
			totalDist += dist;
			separa += calSeparation(my_pos - their_pos[i]);
			cohes += calCohesion(their_pos[i]) * their_pos[i].w;
			totalMass += their_pos[i].w;
		}
	}

	if(totalDist >= DBL_EPSILON)
		align /= totalDist;

	if(totalMass >= DBL_EPSILON)
		cohes = cohes / totalMass - glm::vec3(my_pos);

	//if(glm::length(align) < DBL_EPSILON  && glm::length(separa) < DBL_EPSILON && glm::length(cohes) < DBL_EPSILON)
	//	return glm::vec3(10,0,0);

	return (float)ALIGNMENT * align + (float)SEPARATION * separa + (float)COHESION * cohes;
}


//Simple Euler integration scheme
__global__
void updateDroid(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec4 * pre_pos, int P)
{
	//extern __shared__ glm::vec4 shPos[blockSize];  
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
    if( index < N )
    {
		glm::vec4 my_pos = pos[index];
		float scale = 1;
		int range = RANGE;
		glm::vec3 dir;

		bool isPredator = false;
		for(int i = 0; i < P; i++)
		{
			float len = glm::length(my_pos - pre_pos[i]);
			if(len < RNEIGHBOR*4 && len > DBL_EPSILON)
			{
				dir = glm::normalize(glm::vec3(my_pos) - glm::vec3(pre_pos[i]));
				scale = 10;
				isPredator = true;
				break;
			}
		}

		if(!isPredator){
			if(my_pos.x > range || my_pos.y > range || my_pos.z > range || my_pos.x < -range || my_pos.y < -range || my_pos.z < -range){
				scale = 5;
				dir = glm::normalize(glm::vec3(0,0,0) - glm::vec3(my_pos));
			}
			else{
				dir = FLOCK(N, my_pos, pos, vel);
				if(glm::length(dir) > DBL_EPSILON)
					dir = glm::normalize(dir);
			}
		}

		vel[index] = scale * dir * 5.0f;		
        pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;
    }
}

__global__
void updatePredator(int P, float dt, glm::vec4 * pos, glm::vec3 * vel, int time, glm::vec3 vWander)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
    if( index < P )
    {
		glm::vec4 my_pos = pos[index];
		int range = RANGE;
		glm::vec3 dir;
		float scale = 1;
		if(my_pos.x > range || my_pos.y > range || my_pos.z > range || my_pos.x < -range || my_pos.y < -range || my_pos.z < -range){
			scale = 5;
			glm::vec3 ran = generateRandomNumberFromThread(time, index);
			dir = glm::normalize(glm::vec3(ran.x *scale*scale, ran.y*scale*scale, ran.z *scale*scale) - glm::vec3(my_pos));
		}
		else
		{			
			dir = glm::normalize(vel[index]);
		}
		vel[index] = scale * dir * 6.0f;
		pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;
	}
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__
void sendToVBO(int N, glm::vec4 * pos, glm::vec3 * vel, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<N)
    {
      	float scale = 2.0f;
		glm::vec3 n;
		if(glm::length(vel[index]) > DBL_EPSILON)
		{
			n = glm::normalize(vel[index]);
		}
			/*glm::vec3 up = glm::vec3(0,0,1);

			glm::vec3 right = (glm::cross(up, n));
			if(glm::length(right) > DBL_EPSILON)
				right = glm::normalize(right);
			
			up = (glm::cross(n, right));
			if(glm::length(up) > DBL_EPSILON)
				up = glm::normalize(up);

			glm::mat4x4 modelMatrix;
			modelMatrix[0][0] = right.x;
			modelMatrix[1][0] = right.y;
			modelMatrix[2][0] = right.z;
			modelMatrix[3][0] = pos[index].x;
			
			modelMatrix[0][1] = up.x;
			modelMatrix[1][1] = up.y;
			modelMatrix[2][1] = up.z;
			modelMatrix[3][1] = pos[index].y;

			modelMatrix[0][2] = n.x;
			modelMatrix[1][2] = n.y;
			modelMatrix[2][2] = n.z;
			modelMatrix[3][2] = pos[index].z;

			modelMatrix[0][3] = modelMatrix[1][3] = modelMatrix[2][3] = 0;
			modelMatrix[3][3] = 1;		

			for(int i = 0; i < 12; i += 4)
			{
				glm::vec4 newPos(vbo[12*index+i] / c_scale_w , vbo[12*index+i+1] / c_scale_w, vbo[12*index+i+2] / c_scale_w, vbo[12*index+i+3]);
				newPos = modelMatrix * newPos;
				vbo[12*index+i] = newPos.x * c_scale_w;
				vbo[12*index+i+1] = newPos.y * c_scale_w;
				vbo[12*index+i+2] = newPos.z * c_scale_w;
				vbo[12*index+i+3] = 1;
			}
		}*/


		/*glm::mat4x4 modelMatrix;
		modelMatrix[0][0] = 1;
		modelMatrix[1][0] = 0;
		modelMatrix[2][0] = 0;
		modelMatrix[3][0] = pos[index].x;
			
		modelMatrix[0][1] = 0;
		modelMatrix[1][1] = 1;
		modelMatrix[2][1] = 0;
		modelMatrix[3][1] = pos[index].y;

		modelMatrix[0][2] = 0;
		modelMatrix[1][2] = 0;
		modelMatrix[2][2] = 1;
		modelMatrix[3][2] = pos[index].z;

		modelMatrix[0][3] = modelMatrix[1][3] = modelMatrix[2][3] = 0;
		modelMatrix[3][3] = 1;		

		for(int i = 0; i < 12; i += 4)
		{
			glm::vec4 newPos(vbo[12*index+i] , vbo[12*index+i+1], vbo[12*index+i+2], vbo[12*index+i+3]);
			newPos = modelMatrix * newPos;
			vbo[12*index+i] = newPos.x * c_scale_w;
			vbo[12*index+i+1] = newPos.y * c_scale_w;
			vbo[12*index+i+2] = newPos.z * c_scale_w;
			vbo[12*index+i+3] = 1;
		}*/
	
		
		vbo[12*index+0] = (pos[index].x + 5*scale*n.x)*c_scale_w;
		vbo[12*index+1] = (pos[index].y + 5*scale*n.y)*c_scale_h;
		vbo[12*index+2] = (pos[index].z + 5*scale*n.z)*c_scale_h;
		vbo[12*index+3] = 1;

		vbo[12*index+4] = (pos[index].x - scale*n.x - scale)*c_scale_w;
		vbo[12*index+5] = (pos[index].y - scale*n.y - scale)*c_scale_h;
		vbo[12*index+6] = (pos[index].z - scale*n.z - scale)*c_scale_h;
		vbo[12*index+7] = 1;

		vbo[12*index+8] = (pos[index].x - scale*n.x + scale)*c_scale_w;
		vbo[12*index+9] = (pos[index].y - scale*n.y + scale)*c_scale_h;
		vbo[12*index+10] = (pos[index].z - scale*n.z + scale)*c_scale_h;
		vbo[12*index+11] = 1;
	}
}



__global__
void sendToVBOPre(int P, glm::vec4 * pos, glm::vec3 * vel, float * vbo, int width, int height, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<P)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
		vbo[4*index+2] = pos[index].z*c_scale_w;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__
void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = index % width;
    int y = index / width;
    float w2 = width / 2.0;
    float h2 = height / 2.0;

    float c_scale_w = width / s_scale;
    float c_scale_h = height / s_scale;
	

    if(x<width && y<height)
    {
	    glm::vec3 color(0.05, 0.15, 0.3);
        glm::vec3 acc = ACC(N, glm::vec4((x-w2)/c_scale_w,(y-h2)/c_scale_h,0,1), pos);
        float mag = sqrt(sqrt(acc.x*acc.x + acc.y*acc.y + acc.z*acc.z));
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = (mag < 1.0f) ? mag : 1.0f;
    }
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N, int P)
{
    numObjects = N;
	numPredator = P;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize)));

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");	

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");

	generateRandomVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, 5);
	checkCUDAErrorWithLine("Kernel failed!");

	
	fullBlocksPerGrid = dim3((int)ceil(float(P)/float(blockSize)));

	hipMalloc((void**)&pre_pos, P*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&pre_vel, P*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");

	generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(3, numPredator, pre_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");

	generateRandomVelArray<<<fullBlocksPerGrid, blockSize>>>(4, numPredator, pre_vel, 5);
	checkCUDAErrorWithLine("Kernel failed!");

	glm::vec3 ran = generateRandomNumberFromThread(6, P);
	float theta = ran.x * TWO_PI;
	float u = ran.y * 2.f - 1.f;
	vWander = glm::vec3 (cos(theta)*sqrtf(1-u*u), glm::sin(theta)*sqrtf(1-u*u), u) * kWander;
}

void cudaNBodyUpdateWrapper(float dt, int time)
{    
	dim3 fullBlocksPerGrid((int)ceil(float(numPredator)/float(blockSize)));
	updatePredator<<<fullBlocksPerGrid, blockSize>>>(numPredator, dt, pre_pos, pre_vel, time, vWander);
    checkCUDAErrorWithLine("Kernel failed!");
	
	fullBlocksPerGrid = dim3((int)ceil(float(numObjects)/float(blockSize)));
	updateDroid<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel, pre_pos, numPredator);
    checkCUDAErrorWithLine("Kernel failed!");	
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    sendToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel, vbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");	  
}

void cudaUpdateVBOPre(float * vbodptr, int width, int height)
{
	dim3 fullBlocksPerGrid((int)ceil(float(numPredator)/float(blockSize)));
    sendToVBOPre<<<fullBlocksPerGrid, blockSize>>>(numPredator, pre_pos, pre_vel, vbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");	  
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
	dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");	
}
