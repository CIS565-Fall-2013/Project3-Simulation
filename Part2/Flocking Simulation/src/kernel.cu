#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

#if SHARED == 1
    #define ACC(x,y,z) sharedMemAcc(x,y,z)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
#endif

//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
const float boidMass = 3e8;
const __device__ float starMass = 5e10;

const float scene_scale = 2e2; //size of the height map in simulation space
const __device__ float neighborRadius = 8.0f;
const __device__ float g_fVelKv = 0.5f; 
const __device__ float g_fMaxSpeed = 4.0f;
const __device__ float g_fMaxAccel = 10.0f;
const __device__ float neighborAngle = 180.0f;

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		getchar();
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Generate randomized starting positions for the planets in the XY plane
//Also initialized the masses
__global__ void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0f;//rand.z;
        arr[index].w = mass;
    }
}

//Determine velocity from the distance from the center star. Not super physically accurate because 
//the mass ratio is too close, but it makes for an interesting looking scene
__global__ void generateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec4 * pos)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G*starMass/r);
        glm::vec3 D = glm::normalize(glm::cross(R/r,glm::vec3(0,0,1)));
        arr[index].x = s*D.x;
        arr[index].y = s*D.y;
        arr[index].z = s*D.z;
    }
}

//Generate randomized starting velocities in the XY plane
__global__ void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index) - 0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0;//rand.z;
    }
}

//TODO: Determine force between two bodies
__device__ glm::vec3 calculateAcceleration(glm::vec4 us, glm::vec4 them)
{
    //    G*m_us*m_them
    //F = -------------
    //         r^2
    //
    //    G*m_us*m_them   G*m_them
    //a = ------------- = --------
    //      m_us*r^2        r^2
	glm::vec3 usPosition(us.x, us.y, us.z);
	glm::vec3 themPosition(them.x, them.y, them.z);
	glm::vec3 gravityDir = themPosition - usPosition;
	float _1overR = abs(glm::length(gravityDir)) < EPSILON ? 0 : 1.0f / glm::length(gravityDir);

    return gravityDir * (float)G * them.w * _1overR*_1overR*_1overR;
}

//TODO: Core force calc kernel global memory
__device__  glm::vec3 naiveAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
    glm::vec3 acc = calculateAcceleration(my_pos, glm::vec4(0,0,0,starMass));
	for(int i = 0; i < N; ++i) // read global memory N times
	{
		acc += calculateAcceleration(my_pos, their_pos[i]);
	}
//	printf("acc.x = %f, acc.y = %f, acc.z = %f\n", acc.x, acc.y, acc.z);
    return acc;
}


//TODO: Core force calc kernel shared memory
__device__ glm::vec3 sharedMemAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
    glm::vec3 acc = calculateAcceleration(my_pos, glm::vec4(0,0,0,starMass));

	__shared__ glm::vec4 sharedPositions[blockSize];
//	if((threadIdx.x + (blockIdx.x * blockDim.x)) == 0) printf("%d\n",(int)ceil((float)N / blockSize));
	int positionsFullBlocks = (int)ceil((float)N / blockSize);
	for(int i = 0; i < positionsFullBlocks; ++i)
	{
		int index = threadIdx.x + i * blockDim.x;
/*
		if(index < N)
		{
			sharedPositions[threadIdx.x] = their_pos[index];		
		}
		__syncthreads();
		for(int j = 0; j < blockSize; ++j) 
		{
			acc += calculateAcceleration(my_pos, sharedPositions[j]);
		}
		__syncthreads();*/
		if(index < N)
		{
			sharedPositions[threadIdx.x] = their_pos[index];		
			__syncthreads();
			for(int j = 0; j < blockSize; ++j) 
			{
				acc += calculateAcceleration(my_pos, sharedPositions[j]);
			}
			__syncthreads();
		}
		else
			__syncthreads();
	}
//	printf("acc.x = %f, acc.y = %f, acc.z = %f\n", acc.x, acc.y, acc.z);
    return acc;
}


//Integration 
__global__ void update(int N, float dt, glm::vec4 * pos, glm::vec3 * vel)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
		glm::vec3 myPosition(pos[index].x, pos[index].y, pos[index].z);
		glm::vec3 myVelocity = vel[index];

		int numberOfNeighbors = 0;
		glm::vec3 alignmentNumerator(0.0f);	
		glm::vec3 alignmentVelocity(0.0f);
		glm::vec3 separationVel(0.0f);
		glm::vec3 centerOfMas(0.0f);
		// find neighborhood
		for(int i = 0; i < N; ++i) 
		{
			glm::vec3 theirPos(pos[i].x, pos[i].y, pos[i].z);
			float distanceToNeighbor = glm::distance(myPosition, theirPos) + EPSILON;
			if(distanceToNeighbor < neighborRadius && glm::dot(glm::normalize(myVelocity), glm::normalize(theirPos - myPosition)) > cos(neighborAngle/2))
			{
				alignmentNumerator += vel[i];				
				separationVel += (myPosition - theirPos) / distanceToNeighbor /distanceToNeighbor;
				centerOfMas += theirPos;
				++numberOfNeighbors;

			}
		}
		alignmentVelocity = numberOfNeighbors > 0 ? (alignmentNumerator / float(numberOfNeighbors)) : myVelocity;
		centerOfMas = numberOfNeighbors > 0 ? (centerOfMas / float(numberOfNeighbors)) : myPosition;
		glm::vec3 desiredVel = alignmentVelocity + 0.5f*separationVel + 0.05f*(centerOfMas - myPosition);
		// calculate allignment velocity

		// calculate cohesion velocity

		// calculate separation velocity

//        glm::vec3 acc = ACC(N, my_pos, pos);

		vel[index] += g_fVelKv * (desiredVel - myVelocity) * dt;

    }
}

__global__ void updatePosition(int N, float dt, glm::vec4 * pos, glm::vec3 * vel)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        // RK4 method
		glm::vec3 k1 = vel[index];
		glm::vec3 k2 = k1 + 0.5f * dt * k1;
		glm::vec3 k3 = k1 + 0.5f * dt * k2;
		glm::vec3 k4 = k1 + dt * k3;

		glm::vec3 increment = 1.0f/6.0f * (k1 + 2.0f*k2 + 2.0f*k3 + k4);

		pos[index].x += increment.x * dt;
        pos[index].y += increment.y * dt;
        pos[index].z += increment.z * dt;

		 //Euler method
        /*pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;*/
    }
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__ void sendToVBO(int N, glm::vec4 * pos, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<N)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
        vbo[4*index+2] = 0;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__ void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = index % width;
    int y = index / width;
    float w2 = width / 2.0;
    float h2 = height / 2.0;

    float c_scale_w = width / s_scale;
    float c_scale_h = height / s_scale;

    if(x<width && y<height)
    {
        glm::vec3 color(0.05, 0.15, 0.3);
        glm::vec3 acc = ACC(N, glm::vec4((x-w2)/c_scale_w,(y-h2)/c_scale_h,0,1), pos);
        float mag = sqrt(sqrt(acc.x*acc.x + acc.y*acc.y + acc.z*acc.z));
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = (mag < 1.0f) ? mag : 1.0f;
    }
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N)
{
    numObjects = N;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize))); // one dimensional grid

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, boidMass); // one dimensional block
    checkCUDAErrorWithLine("Kernel failed!");
//    generateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
	generateRandomVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, scene_scale/100.0);
    checkCUDAErrorWithLine("Kernel failed!");
}

void cudaFlockingUpdateWrapper(float dt)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    update<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel);
    checkCUDAErrorWithLine("Kernel failed!");
	updatePosition<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel);
	checkCUDAErrorWithLine("Kernel failed!");
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    sendToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");
}


