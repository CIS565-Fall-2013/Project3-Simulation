#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

#if SHARED == 1
    #define ACC(x,y,z) sharedMemAcc(x,y,z)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
#endif

//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
const float planetMass = 3e8;
const __device__ float starMass = 5e10;
const __device__ float GravConst = 6.67384e-11;
const float scene_scale = 2e2; //size of the height map in simulation space

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__
unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ 
glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Generate randomized starting positions for the planets in the XY plane
//Also initialized the masses
__global__
void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0f;//rand.z;
        arr[index].w = mass;
    }
}

//Determine velocity from the distance from the center star. Not super physically accurate because 
//the mass ratio is too close, but it makes for an interesting looking scene
__global__
void generateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec4 * pos)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G*starMass/r);
        glm::vec3 D = glm::normalize(glm::cross(R/r,glm::vec3(0,0,1)));
        arr[index].x = s*D.x;
        arr[index].y = s*D.y;
        arr[index].z = s*D.z;
    }
}

//Generate randomized starting velocities in the XY plane
__global__
void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index) - 0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0;//rand.z;
    }
}

//TODO: Done!
__device__
glm::vec3 calculateAcceleration(glm::vec4 us, glm::vec4 them)
{
    //    G*m_us*m_them
    //F = -------------
    //         r^2
    //
    //    G*m_us*m_them   G*m_them
    //a = ------------- = --------
    //      m_us*r^2        r^2
	glm::vec3 forceDir = glm::vec3 (them.x - us.x, them.y - us.y, them.z - us.z);
	float dist = sqrt (glm::dot (forceDir, forceDir));
	forceDir /= dist;	// Force direction is now normalized and we have distance between the two objects (r)!

	float accVal = (GravConst * them.w) / (dist*dist);
    
    return forceDir * accVal;
}

//TODO: Done!
__device__ 
glm::vec3 naiveAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	// NOTE: their_pos is a pointer to global memory.
	glm::vec3 acc = glm::vec3 (0);
	for (int i = 0; i < N; i ++)
	{	
		if (their_pos [i] == my_pos)
			continue;
		acc += calculateAcceleration(my_pos, their_pos [i]);
	}
	acc += calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));
	return acc;
}


//TODO: Done. NEED TO FIX CRASH WHEN VISUALIZING.
__device__ 
glm::vec3 sharedMemAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	extern __shared__ glm::vec4 shared_pos [];
	
	glm::vec3 acc = glm::vec3 (0);

	// Loop over each block.
	// This ensures that the whole global memory is loaded into the shared memory, one block at a time.
	for (int j = 0; j < gridDim.x; j ++)
	{
		int blockIndex = blockIdx.x + j;
		
		// If trying to load a block beyond the grid boundary, wrap around.
		if (blockIndex >= gridDim.x)
			blockIndex -= gridDim.x;

		// Calculate global memory index that should be accessed by the thread.
		int index = blockDim.x * blockIndex + threadIdx.x;
		// Load the value from global to shared. 
		if (index < N)
			shared_pos [threadIdx.x] = their_pos [index];
		__syncthreads ();

		// Loop over each object, and calculate acceleration.
		for (int i = 0; i < blockDim.x; i ++)
		{	
			if (shared_pos [i] == my_pos)
				continue;
			acc += calculateAcceleration(my_pos, shared_pos [i]);
		}
	}

	acc += calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));
	return acc;
}


//Simple Euler integration scheme
__global__
void update(int N, float dt, glm::vec4 * pos, glm::vec3 * vel)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        glm::vec4 my_pos = pos[index];
        glm::vec3 acc = ACC(N, my_pos, pos);
        vel[index] += acc * dt;
        pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;
    }
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__
void sendToVBO(int N, glm::vec4 * pos, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<N)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
        vbo[4*index+2] = 0;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__
void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = index % width;
    int y = index / width;
    float w2 = width / 2.0;
    float h2 = height / 2.0;

    float c_scale_w = width / s_scale;
    float c_scale_h = height / s_scale;

    if(x<width && y<height)
    {
        glm::vec3 color(0.05, 0.15, 0.3);
        glm::vec3 acc = ACC(N, glm::vec4((x-w2)/c_scale_w,(y-h2)/c_scale_h,0,1), pos);
        float mag = sqrt(sqrt(acc.x*acc.x + acc.y*acc.y + acc.z*acc.z));
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = (mag < 1.0f) ? mag : 1.0f;
    }
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N)
{
    numObjects = N;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize)));

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");
    generateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
    checkCUDAErrorWithLine("Kernel failed!");
}

void cudaNBodyUpdateWrapper(float dt)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    update<<<fullBlocksPerGrid, blockSize, blockSize>>>(numObjects, dt, dev_pos, dev_vel);
    checkCUDAErrorWithLine("Kernel failed!");
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    sendToVBO<<<fullBlocksPerGrid, blockSize, blockSize>>>(numObjects, dev_pos, vbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    checkCUDAErrorWithLine("Kernel failed!");
}
