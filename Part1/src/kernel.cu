#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

#if SHARED == 1
    #define ACC(x,y,z) sharedMemAcc(x,y,z)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
#endif

//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
const float planetMass = 3e8;
const __device__ float starMass = 5e10;

const float scene_scale = 2e2; //size of the height map in simulation space

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;
glm::vec3 * dev_acc;

//FOR RK4
#if RK4==1
glm::vec4* dev_temppos;
glm::vec3* dev_tempvel;
glm::vec3* dev_accumvel;
glm::vec3* dev_accumaccel;
#endif

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__
unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ 
glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Generate randomized starting positions for the planets in the XY plane
//Also initialized the masses
__global__
void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0f;//rand.z;
        arr[index].w = mass;
    }
}

//Determine velocity from the distance from the center star. Not super physically accurate because 
//the mass ratio is too close, but it makes for an interesting looking scene
__global__
void generateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec4 * pos)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G*starMass/r);
        glm::vec3 D = glm::normalize(glm::cross(R/r,glm::vec3(0,0,1)));
        arr[index].x = s*D.x;
        arr[index].y = s*D.y;
        arr[index].z = s*D.z;
    }
}

//Generate randomized starting velocities in the XY plane
__global__
void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index) - 0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0;//rand.z;
    }
}

//TODO: Determine force between two bodies
__device__
glm::vec3 calculateAcceleration(glm::vec4 us, glm::vec4 them)
{
    //    G*m_us*m_them
    //F = -------------
    //         r^2
    //
    //    G*m_us*m_them   G*m_them
    //a = ------------- = --------
    //      m_us*r^2        r^2
    
	glm::vec3 usPos = glm::vec3(us.x,us.y,us.z);
	glm::vec3 themPos = glm::vec3(them.x,them.y,them.z);

	float distSq = 1e-30 + (usPos.x-themPos.x)*(usPos.x-themPos.x) 
		                 + (usPos.y-themPos.y)*(usPos.y-themPos.y)
						 + (usPos.z-themPos.z)*(usPos.z-themPos.z);

	return float(G)*them.w/distSq * (themPos - usPos) / sqrt(distSq);
	
}

//TODO: Core force calc kernel global memory
__device__ 
glm::vec3 naiveAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	glm::vec3 acc = calculateAcceleration(my_pos, glm::vec4(0,0,0,starMass));
	//glm::vec3 acc(0.0f);
	for(int i=0; i<N; ++i)
	{
		acc+= calculateAcceleration(my_pos, their_pos[i]);
	}
    return acc;
}


//TODO: Core force calc kernel shared memory
__device__ 
glm::vec3 sharedMemAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	const int tileSize = blockSize;
	float fractionTiles = (float)N/tileSize;

	int numberOfTiles =  fractionTiles -(int)fractionTiles>0?(int)fractionTiles+1:(int)fractionTiles;
	glm::vec3 acc = calculateAcceleration(my_pos, glm::vec4(0,0,0,starMass));
	
	extern __shared__ glm::vec4 sharedPos[];
	for(int tile=0; tile<numberOfTiles; ++tile)
	{
		int index = tile*tileSize+threadIdx.x;
		if(index<N)
		{
			sharedPos[threadIdx.x] = their_pos[index];
		}
		__syncthreads();

		for(int i=0;i<tileSize;i++)
		{
			acc+= calculateAcceleration(my_pos, sharedPos[i]);
		}
		__syncthreads();
	}
    return acc;
}

//Simple Euler integration scheme
__global__
void updateF(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    glm::vec4 my_pos;
    glm::vec3 accel;

    if(index < N) my_pos = pos[index];

    accel = ACC(N, my_pos, pos);

    if(index < N) acc[index] = accel;
}

__global__
void updateS(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        vel[index]   += acc[index]   * dt;
        pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;
    }
}

__global__
void RK4Step(int N, float rkStep, float accumMultiplier,glm::vec4* pos,glm::vec3* vel,glm::vec3* accel, glm::vec4* tempPos, glm::vec3* tempVel, glm::vec3* accumVel, glm::vec3*  accumAccel)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        glm::vec4 my_pos = tempPos[index];
		glm::vec3 acc = accel[index];
		accumAccel[index] += accumMultiplier*acc;
		accumVel[index] += accumMultiplier*tempVel[index];

		tempVel[index] = vel[index] + rkStep* acc;
        tempPos[index].x = pos[index].x + tempVel[index].x * rkStep;
		tempPos[index].y = pos[index].y + tempVel[index].y * rkStep;
        tempPos[index].z = pos[index].z + tempVel[index].z * rkStep;		
	}
}

__global__
void RK4FinalUpdate(int N, float dt,glm::vec4* pos,glm::vec3* vel, glm::vec3* accumVel, glm::vec3* accumAccel)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
		float oneSixthInverse = dt/6.0f;
        vel[index]   += oneSixthInverse*accumAccel[index];
        pos[index].x += oneSixthInverse*accumVel[index].x;
		pos[index].y += oneSixthInverse*accumVel[index].y;
		pos[index].z += oneSixthInverse*accumVel[index].z;
	}
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__
void sendToVBO(int N, glm::vec4 * pos, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<N)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
        vbo[4*index+2] = 0;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__
void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = index % width;
    int y = index / width;
    float w2 = width / 2.0;
    float h2 = height / 2.0;

    float c_scale_w = width / s_scale;
    float c_scale_h = height / s_scale;

    glm::vec3 color(0.05, 0.15, 0.3);
    glm::vec3 acc = ACC(N, glm::vec4((x-w2)/c_scale_w,(y-h2)/c_scale_h,0,1), pos);

    if(x<width && y<height)
    {
        float mag = sqrt(sqrt(acc.x*acc.x + acc.y*acc.y + acc.z*acc.z));
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = (mag < 1.0f) ? mag : 1.0f;
    }
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N)
{
    numObjects = N;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize)));

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_acc, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");
    generateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
    checkCUDAErrorWithLine("Kernel failed!");

#if RK4 == 1
	//FOR RK4
    hipMalloc((void**)&dev_temppos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_tempvel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_accumaccel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_accumvel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
#endif
    hipDeviceSynchronize();
}

void cudaNBodyUpdateWrapper(float dt)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
#if RK4 == 1
	hipMemcpy( dev_temppos, dev_pos, numObjects*sizeof(glm::vec4),hipMemcpyDeviceToDevice);
	hipMemcpy( dev_tempvel, dev_vel, numObjects*sizeof(glm::vec3),hipMemcpyDeviceToDevice);
	//RK4
	//First step
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_temppos, dev_vel, dev_acc);
	RK4Step<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt/2.0f, 1.0f, dev_pos, dev_vel,dev_acc,dev_temppos,dev_tempvel,dev_accumvel,dev_accumaccel);
	//Second step
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_temppos, dev_vel, dev_acc);
	RK4Step<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt/2.0f, 2.0f, dev_pos, dev_vel,dev_acc,dev_temppos,dev_tempvel,dev_accumvel,dev_accumaccel);
	//Third step
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_temppos, dev_vel, dev_acc);
	RK4Step<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, 2.0f, dev_pos, dev_vel,dev_acc,dev_temppos,dev_tempvel,dev_accumvel,dev_accumaccel);
	//Fourth step
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_temppos, dev_vel, dev_acc);
	RK4Step<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, 1.0f, dev_pos, dev_vel,dev_acc,dev_temppos,dev_tempvel,dev_accumvel,dev_accumaccel);

	RK4FinalUpdate<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel,dev_accumvel,dev_accumaccel);
	hipMemset(dev_accumaccel,0,numObjects*sizeof(glm::vec3));
	hipMemset(dev_accumvel,0,numObjects*sizeof(glm::vec3));
#else
    updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
    checkCUDAErrorWithLine("Kernel failed!");
    updateS<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
	checkCUDAErrorWithLine("Kernel failed!");
#endif
    hipDeviceSynchronize();
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    sendToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}


