#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "glm/glm.hpp"
#include "utilities.h"
#include "kernel.h"

//#if SHARED == 1
//	#define ACC(x,y,z) pfSharedMemAcc(x,y,z)
//	#define FLOCK(p,q,r,s,t) FlockShared(p,q,r,s,t)
//#else
//    #define ACC(x,y,z) naiveAcc(x,y,z)
//	#define FLOCK(p,q,r,s,t) FlockGlobal(p,q,r,s,t)
//#endif

//GLOBALS
dim3 threadsPerBlock(blockSize);

int numObjects;
const float planetMass = 3e8;
const __device__ float starMass = 5e10;
const __device__ float GravConst = 6.67384e-11;
__device__ bool prefetch;
const float scene_scale = 2e2; //size of the height map in simulation space

#if SHARED == 1
//	if (prefetch)
		#define ACC(x,y,z) pfSharedMemAcc(x,y,z)
//	else
//		#define ACC(x,y,z) sharedMemAcc(x,y,z)
	#define FLOCK(p,q,r,s,t) FlockGlobal(p,q,r,s,t)
#else
    #define ACC(x,y,z) naiveAcc(x,y,z)
	#define FLOCK(p,q,r,s,t) FlockGlobal(p,q,r,s,t)
#endif

glm::vec4 * dev_pos;
glm::vec3 * dev_vel;
glm::vec3 * dev_acc;

void checkCUDAError(const char *msg, int line = -1)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		std::cin.get ();
        exit(EXIT_FAILURE); 
    }
} 

__host__ __device__
unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Function that generates static.
__host__ __device__ 
glm::vec3 generateRandomNumberFromThread(float time, int index)
{
    thrust::default_random_engine rng(hash(index*time));
    thrust::uniform_real_distribution<float> u01(0,1);

    return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Generate randomized starting positions for the planets in the XY plane
//Also initialized the masses
__global__
void generateRandomPosArray(int time, int N, glm::vec4 * arr, float scale, float mass)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index)-0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0f;//rand.z;
        arr[index].w = mass;
    }
}

//Determine velocity from the distance from the center star. Not super physically accurate because 
//the mass ratio is too close, but it makes for an interesting looking scene
__global__
void generateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec4 * pos)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G*starMass/r);
        glm::vec3 D = glm::normalize(glm::cross(R/r,glm::vec3(0,0,1)));
        arr[index].x = s*D.x;
        arr[index].y = s*D.y;
        arr[index].z = s*D.z;
    }
}

//Generate randomized starting velocities in the XY plane
__global__
void generateRandomVelArray(int time, int N, glm::vec3 * arr, float scale)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < N)
    {
        glm::vec3 rand = scale*(generateRandomNumberFromThread(time, index) - 0.5f);
        arr[index].x = rand.x;
        arr[index].y = rand.y;
        arr[index].z = 0.0;//rand.z;
    }
}

//TODO: Done!
__device__
glm::vec3 calculateAcceleration(glm::vec4 us, glm::vec4 them)
{
    //    G*m_us*m_them
    //F = -------------
    //         r^2
    //
    //    G*m_us*m_them   G*m_them
    //a = ------------- = --------
    //      m_us*r^2        r^2
	glm::vec3 forceDir = glm::vec3 (them.x - us.x, them.y - us.y, them.z - us.z);
	float dist = sqrt (glm::dot (forceDir, forceDir));
	
	if (dist > 0)
	{
		forceDir /= dist;	// Force direction is now normalized and we have distance between the two objects (r)!
		float accVal = (GravConst * them.w) / (dist*dist);
		return forceDir * accVal;
	}

	return glm::vec3 (0);
}

//TODO: Done!
__device__ 
glm::vec3 naiveAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	// NOTE: their_pos is a pointer to global memory.
	glm::vec3 acc = glm::vec3 (0);
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

//	if (index < N)
//	{	
		for (int i = 0; i < N; i ++)
		{	
			if (their_pos [i] == my_pos)
				continue;
			acc += calculateAcceleration(my_pos, their_pos [i]);
		}
		acc += calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));
//	}
	return acc;
}

// For float comparisons.
__device__ bool isApproximately (const float &a, const float &b)
{
	if ((a >= (b - 0.001)) && (a <= (b + 0.001)))
		return true;
	return false;
}

//TODO: Done! 
__device__ 
glm::vec3 sharedMemAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	extern __shared__ glm::vec4 shared_pos [];
	int threadNo = blockDim.x * blockIdx.x + threadIdx.x;

	glm::vec3 acc = glm::vec3 (0);

//	int loopMax = ceil (N / (float)blockDim.x);

	// Loop over each block (assuming parallelization of objects) and load objects from global to shared memory.
	// The first block of threads will load the first blockDim.x no. of objects from global memory to shared memory; 
	// The next block will load the next blockDim.x no. of objects from global and so on. Thus, we load the entire
	// set of positions in global memory into shared memory iteratively, one block at a time.
	for (int j = 0; j < ceil (N / (float)blockDim.x); j ++)
	{
		// refBlockIndex is the block index of the block of memory locations we're trying to copy into shared.
		int refblockIndex = blockIdx.x + j;
		
		// If trying to load a block beyond the grid boundary, wrap around.
		if (refblockIndex >= ceil (N / (float)blockDim.x))
			refblockIndex -= ceil (N / (float)blockDim.x);

		// Calculate global memory index that should be accessed by this thread.
		int index = blockDim.x * refblockIndex + threadIdx.x;
		// Load the value from global to shared. 
		if (index < N)
			shared_pos [threadIdx.x] = their_pos [index];

//		Synchronize here.
		__syncthreads();		

		// Compute acceleration only for all threads.
		// Loop over each object, and calculate acceleration.
		for (int i = 0; i < blockDim.x; i ++)
		{	
			// If the block of global memory we're loading into shared mem corresponds to the last block in the grid, 
			// it can contain less than blockDim.x elements. In such a situation, break out of the loop once we pass 
			// the last element in that "block".
			if (refblockIndex == (floor (N / (float)blockDim.x)))
				if (i >= (N%blockDim.x))
					break;

			// A body cannot exert a force on itself, so skip..
			if (isApproximately (shared_pos [i].x, my_pos.x) && 
				isApproximately (shared_pos [i].y, my_pos.y) && 
				isApproximately (shared_pos [i].z, my_pos.z))
				continue;

			acc += calculateAcceleration(my_pos, shared_pos [i]);
		}
	}

	// Calculate acceleration due to star.
	acc += calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));
	
	return acc;
}

// Shared memory acceleration calculation with prefetching.
// Written as a separate function to compare performance.
__device__ 
glm::vec3 pfSharedMemAcc(int N, glm::vec4 my_pos, glm::vec4 * their_pos)
{
	extern __shared__ glm::vec4 shared_pos [];
	int threadNo = blockDim.x * blockIdx.x + threadIdx.x;

	glm::vec3 acc = glm::vec3 (0);
	glm::vec4 prefetcher = glm::vec4 (0);

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
		prefetcher = their_pos [index];		// Prefetch first element into register.

	for (int j = 0; j < ceil (N / (float)blockDim.x); j ++)
	{
		int refblockIndex = blockIdx.x + j + 1;
		if (refblockIndex >= ceil (N / (float)blockDim.x))
			refblockIndex -= ceil (N / (float)blockDim.x);

		index = blockDim.x * refblockIndex + threadIdx.x;

		shared_pos [threadIdx.x] = prefetcher;	// Copy prefetched element into shared memory.
		prefetcher = glm::vec4 (0);
		__syncthreads();		

		if (index < N)
			if (j < floor (N / (float)blockDim.x))		// Prefetching to stop at the penultimate block, after the final
				prefetcher = their_pos [index];			// block has been loaded.

		for (int i = 0; i < blockDim.x; i ++)
		{	
			if (refblockIndex == (floor (N / (float)blockDim.x)))
				if (i >= (N%blockDim.x))
					break;

			if (isApproximately (shared_pos [i].x, my_pos.x) && 
				isApproximately (shared_pos [i].y, my_pos.y) && 
				isApproximately (shared_pos [i].z, my_pos.z))
				continue;

			acc += calculateAcceleration(my_pos, shared_pos [i]);
		}
	}

	// Calculate acceleration due to star.
	acc += calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));
	
	return acc;
}

// Calculate flocking velocity.
__device__ glm::vec3 FlockGlobal (int N, float DT, glm::vec4 my_pos, glm::vec4 *pos, glm::vec3 *vel)
{
	glm::vec3	acc = glm::vec3 (0);
	glm::vec3 my_vel;

	glm::vec3 sumVelocities = glm::vec3 (0);
	glm::vec3 sumPositions = glm::vec3 (0);
	glm::vec3 sumSepVelocities = glm::vec3 (0);

	int neighbours = 0;

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N)
	{
		my_vel = vel [index];

		for (int i = 0; i < N; i ++)
		{	
			glm::vec4 curPos = pos [i];
			float distance = glm::length (curPos - my_pos);
			
			if (distance <= 5.0)
			{
				sumVelocities += vel [i];

				sumPositions.x += curPos.x;
				sumPositions.y += curPos.y;
				sumPositions.z += curPos.z;

				sumSepVelocities.x += (my_pos.x - curPos.x);
				sumSepVelocities.y += (my_pos.y - curPos.y);
				sumSepVelocities.z += (my_pos.z - curPos.z);

				neighbours ++;
			}
		}

		if (neighbours > 0)
		{
			sumSepVelocities /= neighbours;
			sumPositions /= neighbours;		// Centre of mass.
			sumVelocities /= neighbours;	
		}

		// Calculate total velocity:
		glm::vec3 flockVel = (safeNormalize (sumVelocities) * glm::length (my_vel))//	* 0.4f							// Align component
							 +	(safeNormalize (sumPositions - glm::vec3 (my_pos)) * glm::length (my_vel))//	* 0.2f	// Cohesion component
							 +	(safeNormalize (sumSepVelocities) * glm::length (my_vel));// * 0.4f;					// Separation component

		acc = ((glm::length (flockVel) - glm::length (my_vel))/DT) * safeNormalize (flockVel);
	}
	return acc;
}

// normalize only if length > 0
inline __device__ glm::vec3 safeNormalize (glm::vec3 vectorToBeNormalized)
{
	float len = glm::length (vectorToBeNormalized);
	if (len > 0.01)
		return vectorToBeNormalized / len;
	return vectorToBeNormalized;
}

// Calculate gravitational acceleration.
__global__
void updateF(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    glm::vec4 my_pos;
    glm::vec3 accel;

    if(index < N) my_pos = pos[index];

    accel = ACC(N, my_pos, pos);

    if(index < N) acc[index] = accel;
}

// Calculate acceleration for Custom Simulation.
__global__
void updateFCustom (int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    glm::vec4 my_pos;
    glm::vec3 accel;

    if(index < N) my_pos = pos[index];

    accel = FLOCK (N, dt, my_pos, pos, vel) + calculateAcceleration (my_pos, glm::vec4 (0, 0, 0, starMass));

    if(index < N) acc[index] = accel;
}

//Simple Euler integration scheme
__global__
void updateS(int N, float dt, glm::vec4 * pos, glm::vec3 * vel, glm::vec3 * acc)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if( index < N )
    {
        vel[index]   += acc[index]   * dt;
        pos[index].x += vel[index].x * dt;
        pos[index].y += vel[index].y * dt;
        pos[index].z += vel[index].z * dt;
    }
}

//Update the vertex buffer object
//(The VBO is where OpenGL looks for the positions for the planets)
__global__
void sendToVBO(int N, glm::vec4 * pos, float * vbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale_w = -2.0f / s_scale;
    float c_scale_h = -2.0f / s_scale;

    if(index<N)
    {
        vbo[4*index+0] = pos[index].x*c_scale_w;
        vbo[4*index+1] = pos[index].y*c_scale_h;
        vbo[4*index+2] = 0;
        vbo[4*index+3] = 1;
    }
}

//Update the texture pixel buffer object
//(This texture is where openGL pulls the data for the height map)
__global__
void sendToPBO(int N, glm::vec4 * pos, float4 * pbo, int width, int height, float s_scale)
{
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = index % width;
    int y = index / width;
    float w2 = width / 2.0;
    float h2 = height / 2.0;

    float c_scale_w = width / s_scale;
    float c_scale_h = height / s_scale;

    glm::vec3 color(0.05, 0.15, 0.3);
    glm::vec3 acc = ACC(N, glm::vec4((x-w2)/c_scale_w,(y-h2)/c_scale_h,0,1), pos);

	if(x<width && y<height)
	{
        float mag = sqrt(sqrt(acc.x*acc.x + acc.y*acc.y + acc.z*acc.z));
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = (mag < 1.0f) ? mag : 1.0f;
    }
}

/*************************************
 * Wrappers for the __global__ calls *
 *************************************/

//Initialize memory, update some globals
void initCuda(int N)
{
    numObjects = N;
    dim3 fullBlocksPerGrid((int)ceil(float(N)/float(blockSize)));

    hipMalloc((void**)&dev_pos, N*sizeof(glm::vec4));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_vel, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");
    hipMalloc((void**)&dev_acc, N*sizeof(glm::vec3));
    checkCUDAErrorWithLine("Kernel failed!");

    generateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("Kernel failed!");
    generateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
    checkCUDAErrorWithLine("Kernel failed!");
    hipDeviceSynchronize();
}

void cudaNBodyUpdateWrapper(float dt, bool customSimulation)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
	if (customSimulation)
		updateFCustom<<<fullBlocksPerGrid, blockSize/*, blockSize*sizeof(glm::vec4)*/>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
	else
		updateF<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
    checkCUDAErrorWithLine("Kernel failed!");
	//glm::vec3 *accn = new glm::vec3 [numObjects];
	//for (int i= 0; i < numObjects; i ++)
	//	accn [i] = glm::vec3 (0);
	//hipMemcpy (accn, dev_acc, sizeof(glm::vec3)*numObjects, hipMemcpyDeviceToHost);
	//int breakHere = -1;
	updateS<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
    checkCUDAErrorWithLine("Kernel failed!");
    hipDeviceSynchronize();
}

void cudaUpdateVBO(float * vbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects)/float(blockSize)));
    sendToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}

void cudaUpdatePBO(float4 * pbodptr, int width, int height)
{
    dim3 fullBlocksPerGrid((int)ceil(float(width*height)/float(blockSize)));
    sendToPBO<<<fullBlocksPerGrid, blockSize, blockSize*sizeof(glm::vec4)>>>(numObjects, dev_pos, pbodptr, width, height, scene_scale);
    hipDeviceSynchronize();
}

void setDevicePrefetch (bool prefetchEnabled)
{
	hipMemcpyToSymbol(HIP_SYMBOL(&prefetch), &prefetchEnabled, sizeof (bool), 0);
}
